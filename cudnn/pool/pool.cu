#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudnn_frontend.h>
#include <driver_functions.h>
#include <stdlib.h>
#include <unistd.h>

#include <iostream>
#include <vector>

#include "../../../tc-benchmark/nvml_tools.cu"

enum Mode { FP32 = 1, FP16_32 = 2, TF32 = 4 };
enum RESAMPLE { MAX = 1, AVG = 2 };

#define POWER

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) \
  { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif

// using data_type = cudnn_frontend::DataType_t::FLOAT;

/**
 * Builds a graph for convolution operation using cuDNN.
 *
 * @param n         The batch size.
 * @param c         The number of input channels.
 * @param h         The height of the input tensor.
 * @param w         The width of the input tensor.
 * @param handle    The cuDNN handle.
 *
 * @return A tuple containing the constructed graph, input tensor, filter
 * tensor, and output tensor.
 */
auto build_graph(int n, int c, int h, int w, RESAMPLE resample,
                 hipdnnHandle_t handle, int window, int stride, int pad,
                 Mode mode) {
  auto graph = std::make_shared<cudnn_frontend::graph::Graph>();
  if (mode & (FP32 | TF32)) {
    graph->set_io_data_type(cudnn_frontend::DataType_t::FLOAT)
        .set_compute_data_type(cudnn_frontend::DataType_t::FLOAT);
  } else if (mode & FP16_32) {
    graph->set_io_data_type(cudnn_frontend::DataType_t::HALF)
        .set_compute_data_type(cudnn_frontend::DataType_t::FLOAT);
  }

  auto input = graph->tensor(cudnn_frontend::graph::Tensor_attributes()
                                 .set_name("input")
                                 .set_dim({n, c, h, w})
                                 .set_stride({c * h * w, 1, c * w, c}));

  auto res_fn = cudnn_frontend::ResampleMode_t::MAXPOOL;
  auto pad_mode = cudnn_frontend::PaddingMode_t::NEG_INF_PAD;

  if (resample & AVG) {
    res_fn = cudnn_frontend::ResampleMode_t::AVGPOOL_INCLUDE_PADDING;
    pad_mode = cudnn_frontend::PaddingMode_t::ZERO_PAD;
  }

  auto res_options = cudnn_frontend::graph::Resample_attributes()
                         .set_is_inference(true)
                         .set_resampling_mode(res_fn)
                         .set_padding_mode(pad_mode)
                         .set_window({window, window})
                         .set_stride({stride, stride})
                         .set_pre_padding({pad, pad})
                         .set_post_padding({pad, pad});

  auto [Y, Index] = graph->resample(input, res_options);

  Y->set_output(true);
  assert(Index == nullptr);

  graph->validate().is_good();

  graph->build_operation_graph(handle).is_good();

  graph->create_execution_plans({cudnn_frontend::HeurMode_t::A}).is_good();

  graph->check_support(handle).is_good();

  auto plan_count = graph->get_execution_plan_count();
  std::cout << "Number of execution plans: " << plan_count << std::endl;

  graph->build_plans(handle).is_good();

  return std::make_tuple(graph, input, Y);
}

int main(int argc, char *argv[]) {
  int n = 1;
  int c = 3;
  int h = 512;
  int w = 512;
  int window = 2;
  int stride = 1;
  int pad = 0;
  RESAMPLE resample = MAX;
  Mode mode = FP32;

  std::thread measuring_thread;
  monitor_args thread_args;
  thread_args.powerArray = std::vector<int>();
  thread_args.clockArray = std::vector<int>();
  thread_args.flag = 0;

  init_nvml(&thread_args, &measuring_thread);
  cudaCheckError(hipDeviceSynchronize());

  srand(0);

  // parse command line arguments, set args for conv
  int arg;
  hipSetDevice(0);
  while ((arg = getopt(argc, argv, "n:c:h:w:f:s:p:MAm:")) != -1) switch (arg) {
      case 'n':
        n = atoi(optarg);
        break;
      case 'c':
        c = atoi(optarg);
        break;
      case 'h':
        h = atoi(optarg);
        break;
      case 'w':
        w = atoi(optarg);
        break;
      case 'f':
        window = atoi(optarg);
        break;
      case 's':
        stride = atoi(optarg);
        break;
      case 'p':
        pad = atoi(optarg);
        break;
      case 'M':
        resample = MAX;
        break;
      case 'A':
        resample = AVG;
        break;
      case 'm':
        mode = static_cast<Mode>(atoi(optarg));
        if (mode != FP32 && mode != FP16_32 && mode != TF32) {
          fprintf(stderr, "Invalid mode\n");
          exit(EXIT_FAILURE);
        }
        break;
      default:
        fprintf(stderr,
                "Usage: %s [OPTION]...\n\n\t-n \t The batch size [int] "
                "[default=1]\n\t-c \t The number of input channels [int] "
                "[default=3]\n\t-h \t The height of the input tensor [int] "
                "[default=512]\n\t-w \t The width of the input tensor [int] "
                "[default=512]\n\t-f \t The window size [int] "
                "[default=3]\n\t-s \t The stride size [int] "
                "[default=1]\n\t-p \t The padding size [int] "
                "[default=0]\n\t-M \t Use MAXPOOL resampling [flag]\n\t-A \t "
                "Use AVGPOOL resampling [flag]\n\t-m \t The mode [int] "
                "[default=1]\n",
                argv[0]);
        exit(EXIT_FAILURE);
    }

  printf("Resample with settings:\n");
  printf("Batch size: %d\n", n);
  printf("Input channels: %d\n", c);
  printf("Input height: %d\n", h);
  printf("Input width: %d\n", w);
  printf("Window: %d\n", window);
  printf("Stride: %d\n", stride);
  printf("Padding: %d\n", pad);
  if (resample & MAX)
    printf("Resample: MAXPOOL\n");
  else if (resample & AVG)
    printf("Resample: AVGPOOL\n");

  long elements_computed = std::floor((h - window + 2 * pad) / stride + 1) *
                           std::floor((w - window + 2 * pad) / stride + 1) * c *
                           n * window * window;

  printf("Elements computed: %ld\n", elements_computed);
  // allocate memory for input, filter, and output tensors
  float *hostInput = (float *)calloc(n * c * h * w, sizeof(float));
  float *hostOutput = (float *)calloc(n * c * h * w, sizeof(float));

  for (int i = 0; i < n * c * h * w; i++) {
    hostInput[i] = ((float)rand() / (float)RAND_MAX) * 2 - 1;
  }

  // allocate memory for input, filter, and output tensors on device
  float *deviceInput, *deviceOutput;
  cudaCheckError(
      hipMalloc((void **)&deviceInput, n * c * h * w * sizeof(float)));
  cudaCheckError(
      hipMalloc((void **)&deviceOutput, n * c * h * w * sizeof(float)));

  // copy input and filter tensors to device
  cudaCheckError(hipMemcpy(deviceInput, hostInput,
                            n * c * h * w * sizeof(float),
                            hipMemcpyHostToDevice));
  cudaCheckError(hipDeviceSynchronize());

  hipdnnHandle_t handle;
  hipdnnCreate(&handle);

  auto [graph, input, Y] =
      build_graph(n, c, h, w, resample, handle, window, stride, pad, mode);

  int8_t *workspace_ptr;
  cudaCheckError(
      hipMalloc((void **)&workspace_ptr, graph->get_workspace_size()));

  std::unordered_map<std::shared_ptr<cudnn_frontend::graph::Tensor_attributes>,
                     void *>
      variant_pack = {{input, deviceInput}, {Y, deviceOutput}};

  std::cout << *graph << std::endl;
  thread_args.flag = 1;
  auto status = graph->execute(handle, variant_pack, workspace_ptr);

#ifdef POWER
#pragma unroll
  for (int i = 0; i < 32768; i++)
    status = graph->execute(handle, variant_pack, workspace_ptr);
#endif

  cudaCheckError(hipDeviceSynchronize());
  thread_args.flag = 0;
  stop_nvml(&measuring_thread, thread_args.powerArray, thread_args.clockArray);

  cudaCheckError(hipDeviceSynchronize());
  std::cout << "Execution status: " << status.get_code() << ":"
            << status.get_message() << std::endl;

  cudaCheckError(hipMemcpy(hostOutput, deviceOutput,
                            n * c * h * w * sizeof(float),
                            hipMemcpyDeviceToHost));

  printf("%f\n", hostOutput[0]);

  // free memory
  cudaCheckError(hipFree(deviceInput));
  cudaCheckError(hipFree(deviceOutput));
  free(hostInput);
  free(hostOutput);

  hipdnnDestroy(handle);
}