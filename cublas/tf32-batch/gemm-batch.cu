/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdlib.h>
#include <unistd.h>

#include <iostream>
#include <vector>

#include "cublas_utils.h"

using data_type = float;

int run(int m, int n, int k, int batch_size, int tensor_core) {
  const int length_m = m;
  const int length_n = n;
  const int length_k = k;
  const int batch = batch_size;

  std::vector<std::vector<data_type>> A(
      batch, std::vector<data_type>(length_m * length_k, 1.0 / batch));
  std::vector<std::vector<data_type>> B(
      batch, std::vector<data_type>(length_k * length_n, 1.0 / batch));
  std::vector<std::vector<data_type>> C(
      batch, std::vector<data_type>(length_m * length_n, 0.0));

  const data_type alpha = 1.0;
  const data_type beta = 0.0;

  data_type **d_A_array = nullptr;
  data_type **d_B_array = nullptr;
  data_type **d_C_array = nullptr;

  std::vector<data_type *> d_A(batch, nullptr);
  std::vector<data_type *> d_B(batch, nullptr);
  std::vector<data_type *> d_C(batch, nullptr);

  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  hipblasHandle_t handle;
  CUBLAS_CHECK(hipblasCreate(&handle));

  if (tensor_core != 0)
    // Set math mode to allow TF32 tensor core operations
    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH));

  /* step 2: copy data to device */
  for (int i = 0; i < batch; i++) {
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A[i]),
                          sizeof(data_type) * A[i].size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B[i]),
                          sizeof(data_type) * B[i].size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C[i]),
                          sizeof(data_type) * C[i].size()));
  }

  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A_array),
                        sizeof(data_type *) * batch));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B_array),
                        sizeof(data_type *) * batch));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_array),
                        sizeof(data_type *) * batch));

  for (int i = 0; i < batch; i++) {
    CUDA_CHECK(hipMemcpy(d_A[i], A[i].data(), sizeof(data_type) * A[i].size(),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B[i], B[i].data(), sizeof(data_type) * B[i].size(),
                          hipMemcpyHostToDevice));
  }

  CUDA_CHECK(hipMemcpy(d_A_array, d_A.data(), sizeof(data_type *) * batch,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B_array, d_B.data(), sizeof(data_type *) * batch,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_C_array, d_C.data(), sizeof(data_type *) * batch,
                        hipMemcpyHostToDevice));

  CUBLAS_CHECK(hipblasSgemmBatched(
      handle, transa, transb, length_m, length_m, length_k, &alpha, d_A_array,
      length_k, d_B_array, length_k, &beta, d_C_array, length_m, batch));

  CUDA_CHECK(hipDeviceSynchronize());

  for (int i = 0; i < batch; i++) {
    CUDA_CHECK(hipMemcpy(C[i].data(), d_C[i], sizeof(data_type) * C[i].size(),
                          hipMemcpyDeviceToHost));
  }

  /* free resources */
  CUDA_CHECK(hipFree(d_A_array));
  CUDA_CHECK(hipFree(d_B_array));
  CUDA_CHECK(hipFree(d_C_array));
  for (int i = 0; i < batch; i++) {
    CUDA_CHECK(hipFree(d_A[i]));
    CUDA_CHECK(hipFree(d_B[i]));
    CUDA_CHECK(hipFree(d_C[i]));
  }

  CUBLAS_CHECK(hipblasDestroy(handle));

  CUDA_CHECK(hipDeviceReset());

  printf("C[0][0] = %f\n", C[0][0]);

  return EXIT_SUCCESS;
}

int main(int argc, char *argv[]) {
  int m = 1024;
  int n = 1024;
  int k = 1024;

  int batch_size = 1;
  int tensor_core = 1;

  int c;
  hipSetDevice(0);
  while ((c = getopt(argc, argv, "m:n:k:a:b:ch")) != -1) switch (c) {
      case 'a':
        m = n = k = atoi(optarg);
        break;
      case 'm':
        m = atoi(optarg);
        break;
      case 'n':
        n = atoi(optarg);
        break;
      case 'k':
        k = atoi(optarg);
        break;
      case 'b':
        batch_size = atoi(optarg);
        break;
      case 'c':
        tensor_core = 0;
        break;
      case 'h':
        fprintf(stdout,
                "Usage: %s [OPTION]...\n\n\t-m \t M dimension [int] "
                "[default=1024]\n\t-n \t N "
                "dimension [int] [default=1024]\n\t-k \t K dimension [int] "
                "[default=1024]\n\t-a \t All "
                "dimensions [int]\n\t-b \t Batch Size [int] [default=1]\n\t-c "
                "\t Disable Tensor Cores\n\n",
                argv[0]);
        exit(EXIT_SUCCESS);
      default:
        fprintf(stderr,
                "Usage: %s [OPTION]...\n\n\t-m \t M dimension [int] "
                "[default=1024]\n\t-n \t N "
                "dimension [int] [default=1024]\n\t-k \t K dimension [int] "
                "[default=1024]\n\t-a \t All "
                "dimensions [int]\n\t-b \t Batch Size [int] [default=1]\n\t-c "
                "\t Disable Tensor Cores\n\n\n\n",
                argv[0]);
        exit(EXIT_FAILURE);
    }

  printf("GEMM with dimensions m=%d, n=%d, k=%d\nBatch Size: %d\n", m, n, k,
         batch_size);

  return run(m, n, k, batch_size, tensor_core);
}